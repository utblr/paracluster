#include <stdio.h>
#include <hip/hip_runtime.h>



#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void cuda_hello(){
  if (threadIdx.x == 0) {
    printf("Hello World from GPU!\n");
  }
}

int main() {
    cuda_hello<<<1,1>>>();
    hipError_t cudaerr = hipDeviceSynchronize();
    cudaCheckErrors("kernel fail");
    if (cudaerr != hipSuccess) {
      printf("kernel launch failed with error \"%s\".\n",
	     hipGetErrorString(cudaerr));
    }
    else {
      printf("Successful!\n");
    }

    return 0;
}
