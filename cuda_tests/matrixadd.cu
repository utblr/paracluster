#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 4

__global__ void MatAdd(int A[N*N], int B[N*N], int C[N*N]) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row + col*N;
  printf("idx = %d\n", idx);

  if (row < N && col < N) {
    C[idx] = A[idx] + B[idx];
  }
}

int main() {
  int A[N*N];
  int B[N*N];
  int C[N*N];
  int mysize = N*N*sizeof(int);

  for (int i = 0; i < N*N; i++) {
    A[i] = i;
    B[i] = i;    
  }

  int *cudaA = 0;
  int *cudaB = 0;
  int *cudaC = 0;

  hipMalloc(&cudaA, mysize);
  hipMalloc(&cudaB, mysize);
  hipMalloc(&cudaC, mysize);  

  hipMemcpy(cudaA, A, mysize, hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, mysize, hipMemcpyHostToDevice);
  hipMemcpy(cudaC, C, mysize, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(N,N);

  MatAdd <<< 1, threadsPerBlock >>> (cudaA, cudaB, cudaC);
  hipDeviceSynchronize();

  hipMemcpy(C, cudaC, mysize, hipMemcpyDeviceToHost);
  hipFree(cudaC);

  for (int i = 0; i < N*N; i++) {
    printf("%d ", C[i]);
  }

  printf("\n");
  return 0;
}